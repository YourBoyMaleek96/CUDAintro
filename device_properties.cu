#include "hip/hip_runtime.h"
/*************************************************************************************************************************
* Name: Malik Freeman
* Date: 2/16/2025
* Course: SSE 662 - Design, Maintenance, and Quality
* Assignment: Module 3 Programming Task
* File Name: device_properties.cu
* Description: This file implements a program that retrieves and prints the properties of each available CUDA device.
*              The program uses the CUDA Runtime API to query the number of CUDA devices available on the system and
*              then retrieves the properties of each device. The program prints the device name, compute capability,
*              number of multiprocessors, maximum threads per multiprocessor, maximum threads per block, and clock rate
*              for each device. The program also handles the special case of devices with compute capability (9999, 9999).
*              The program uses the cudasafe function to check CUDA API call results and handle errors.
*
*************************************************************************************************************************/



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
// Helper function to check CUDA API call results and handle errors
void cudasafe(hipError_t error, const char* message, const char* file, int line) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s : %i. In %s line %d\n", message, error, file, line); 
        exit(-1);
    }
}

int main(int argc, char ** argv) {
    int deviceCount; 

    // Check the number of CUDA devices available
    cudasafe(hipGetDeviceCount(&deviceCount), "hipGetDeviceCount", __FILE__, __LINE__); 

    printf("Number of CUDA devices: %d.\n", deviceCount); 

    // Handle the case of no CUDA devices
    if (deviceCount == 0) {
        printf("No CUDA devices found.\n");
        return -1;
    }

    // Loop through each available device and retrieve its properties
    for (int dev = 0; dev < deviceCount; dev++) {
        hipDeviceProp_t deviceProp;

        // Retrieve device properties for each CUDA device
        cudasafe(hipGetDeviceProperties(&deviceProp, dev), "hipGetDeviceProperties", __FILE__, __LINE__);

        // Print detailed properties for the current device
        printf("\nFor device #%d:\n", dev); 
        printf("Device name:                %s\n", deviceProp.name); 
        printf("Compute capability (major): %d\n", deviceProp.major);
        printf("Compute capability (minor): %d\n", deviceProp.minor);        
        printf("Number of multiprocessors:   %d\n", deviceProp.multiProcessorCount);
        printf("Max threads per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
        printf("Max threads per block:      %d\n", deviceProp.maxThreadsPerBlock);
        printf("Clock rate (kHz):           %d\n", deviceProp.clockRate);

        // Handle special case for devices with compute capability (9999, 9999)
        if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
            printf("This device has an unknown compute capability.\n");
        }
    }

    // Pause the system to allow user to see output before exiting (for Windows)
    system("pause");
    return 0;
}
